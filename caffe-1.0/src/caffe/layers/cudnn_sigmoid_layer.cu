
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
      CUDNN_ACTIVATION_SIGMOID,
      this->bottom_desc_, bottom_data, this->top_desc_, top_data));
}

template <typename Dtype>
void CuDNNSigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
      CUDNN_ACTIVATION_SIGMOID,
      this->top_desc_, top_data, this->top_desc_, top_diff,
      this->bottom_desc_, bottom_data, this->bottom_desc_, bottom_diff));
}

INSTANTIATE_CLASS(CuDNNSigmoidLayer);

}  // namespace caffe
#endif
